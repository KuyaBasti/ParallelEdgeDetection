#include "hip/hip_runtime.h"
#include "cuda_image.hpp"

CudaImage::CudaImage(Image &from)
{
    // from parallel_image.hpp
    _x = from.x();
    _y = from.y();
    _type = from.type();
    _isstb = false;

    // Allocate GPU memory
    hipMalloc(&_data, pixel_size() * pixel_count());

    // Copy data to GPU 
    hipMemcpy(_data, from.data(), pixel_size() * pixel_count(), hipMemcpyHostToDevice);
}

CudaImage::CudaImage(unsigned int x, unsigned int y, ImageType type) {
    _x = x;
    _y = y;
    _type = type;
    _isstb = false;
    
    // Allocate GPU memory
    hipMalloc(&_data, pixel_size() * pixel_count());

    // Set memory to 0
    hipMemset(_data, 0, pixel_size() * pixel_count());
}

CudaImage::~CudaImage()
{
    if (_data) {
        hipFree(_data);
        _data = nullptr;
    }
}

std::shared_ptr<Image> CudaImage::to_host()
{
    validate();  
    auto new_image = std::make_shared<Image>(_x, _y, _type);
    hipMemcpy(new_image->data(), _data, pixel_size() * pixel_count(), hipMemcpyDeviceToHost);
    return new_image;
}

                                    // RGB DATA             GRAYSCALE     WIDTH   HEIGHT
__global__ void convertRGBtoGRAYSCALE(unsigned char* srcdata, float* rdata, int x, int y)
{
    // DISCUSSION RICO
    // 1. Calculate thread ID
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    // 2. Check if thread is in bounds
    if (tidx < x && tidy < y)
    {
        // 3. Calculate 1D index
        int i = tidx + tidy * x;
        
        // 4. Do the work - convert RGB to grayscale
        rdata[i] = ((float)srcdata[i * 3] +
                    (float)srcdata[i * 3 + 1] +
                    (float)srcdata[i * 3 + 2]) /
                    (255.0f * 3.0f);
    }
}

__global__ void convertFLOATINGGRAYSCALEtoGRAYSCALE(float* srcdata, unsigned char* rdata, int x, int y)
{   // DISCUSSION RICO
    // 1. Calculate thread ID
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    // 2. Check if thread is in bounds
    if (tidx < x && tidy < y)
    {
        // 3. Calculate 1D index
        int i = tidx + tidy * x;
        
        // 4. Do the work - convert floatingpioint grayscal to grayscale
        rdata[i] = (unsigned char)(srcdata[i] * 255.0f);
    }
}

std::shared_ptr<Image> CudaImage::convert(ImageType to, int mode)
{
    // which conversion we are doing?
    if (mode == 0) {
        // RGB -> floatgrayscale
        if (to == floatgrayscale && _type == rgb) {
            // RGB to GRASCALE conversion
            // 1. Create new CudaImage for result
            // 2. Set up CUDA grid/block dimensions
            // 3. Launch RGB to grayscale kernel
            // 4. Return result
            auto ret = std::make_shared<CudaImage>(_x, _y, to);

            dim3 gridDim(_x/32, _y/32,1);
            dim3 blockDim(32, 32, 1);

            convertRGBtoGRAYSCALE<<<gridDim, blockDim>>>((unsigned char*)_data, (float*)ret->_data, _x, _y);
            
            return ret;
        }
        // floatgrayscale -> grayscale 
        else if (to == grayscale && _type == floatgrayscale) {
            // GRAYSCALE to RGB conversion
            // _data is source which is floatgrayscale
            // ret->_data is destination where we want the RGB result
            auto ret = std::make_shared<CudaImage>(_x, _y, to);
            
            dim3 gridDim(_x/32, _y/32,1);
            dim3 blockDim(32, 32, 1);

            convertFLOATINGGRAYSCALEtoGRAYSCALE<<<gridDim, blockDim>>>((float*)_data, (unsigned char*)ret->_data, _x, _y);
            
            return ret;
        }
    }
    return Image::convert(to);
}

__constant__ float gaussian[5][5] = {
    {0.0002f, 0.0033f, 0.0081f, 0.0033f, 0.0002f},
    {0.0033f, 0.0479f, 0.1164f, 0.0479f, 0.0033f},
    {0.0081f, 0.1164f, 0.2831f, 0.1164f, 0.0081f},
    {0.0033f, 0.0479f, 0.1164f, 0.0479f, 0.0033f},
    {0.0002f, 0.0033f, 0.0081f, 0.0033f, 0.0002f}};

__global__ void blur_kernel(float* srcdata, float* rdata, int x, int y)
{
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (tidx < x && tidy < y)
    {
        float tmp = 0;
        for (int i = -2; i < 3; ++i)
        {
            for (int j = -2; j < 3; ++j)
            {
                int xcoord = tidx + i;
                int ycoord = tidy + j;
                if (xcoord < 0)
                    xcoord = 0;
                if (ycoord < 0)
                    ycoord = 0;
                if (xcoord >= x)
                    xcoord = x - 1;
                if (ycoord >= y)
                    ycoord = y - 1;
                // this is biggest hurdle for performance need to do direct mem access
                // how to get index of 2D image in 1D array bc its how its stored
                // index = row * width + column
                // but fppixel uses x + y * _x where x=column, y=row
                // _x is the width of image
                // _y is height 
                // READ from neighboring pixels
                // (x, y) is at index = x + y * _x in 1D array memory
                float pixel = srcdata[xcoord + ycoord * x];

                tmp += gaussian[i + 2][j + 2] * pixel;
            }
        }
        // WRITE to actual pixel target
        rdata[tidx + tidy * x] = tmp;
    }
    
}

std::shared_ptr<Image> CudaImage::blur(int mode)
{
    if (mode == 0) {
        validate();
        if (_type != floatgrayscale)
        {
            throw ImageException("Currently can only blur on floating point grayscale");
        }
        
        auto ret = std::make_shared<CudaImage>(_x, _y, _type);

        dim3 gridDim(_x/32, _y/32,1);
        dim3 blockDim(32, 32, 1);

        blur_kernel<<<gridDim, blockDim>>>((float*)_data, (float*)ret->_data, _x, _y);
        
        return ret;
    }
    return Image::blur();
}

__constant__ float ydir[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};
__constant__ float xdir[3][3] = {{-1, 0, +1}, {-2, 0, +2}, {-1, 0, +1}};

__global__ void gradient_kernel(float* srcdata, float* rdata, int x, int y)
{
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (tidx < x && tidy < y)
    {
        float xgradient = 0;
        float ygradient = 0;
        for (int i = -1; i < 2; ++i)
        {
            for (int j = -1; j < 2; ++j)
            {
                int xcoord = tidx + i;
                int ycoord = tidy + j;
                if (xcoord < 0)
                    xcoord = 0;
                if (ycoord < 0)
                    ycoord = 0;
                if (xcoord >= x)
                    xcoord = x - 1;
                if (ycoord >= y)
                    ycoord = y - 1;
                // READ from neighboring pixels
                float pixel = srcdata[xcoord + ycoord * x];
                xgradient += xdir[i + 1][j + 1] * pixel;
                ygradient += ydir[i + 1][j + 1] * pixel;
            }
        }
        // WRITE to actual target pixel
       rdata[tidx + tidy * x] = sqrtf(xgradient * xgradient + ygradient * ygradient);
    }
    
}

std::shared_ptr<Image> CudaImage::gradient(int mode)
{
    if (mode == 0) {
        validate();
        if (_type != floatgrayscale)
        {
            throw ImageException("Currently can only gradiant on floating point grayscale");
        }
        
        auto ret = std::make_shared<CudaImage>(_x, _y, _type);

        dim3 gridDim(_x/32, _y/32,1);
        dim3 blockDim(32, 32, 1);

        gradient_kernel<<<gridDim, blockDim>>>((float*)_data, (float*)ret->_data, _x, _y);
       
        return ret;
    }
    return Image::gradient();
}

__global__ void edges_kernel(float* srcdata, float* rdata, int x, int y, float low, float high)
{
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;

    if (tidx < x && tidy < y)
    {
        bool nearstrong = false;
        for (int i = -1; i < 2; ++i)
        {
            for (int j = -1; j < 2; ++j)
            {
                int xcoord = tidx + i;
                int ycoord = tidy + j;
                if (xcoord < 0)
                    xcoord = 0;
                if (ycoord < 0)
                    ycoord = 0;
                if (xcoord >= x)
                    xcoord = x - 1;
                if (ycoord >= y)
                    ycoord = y - 1;
                // READ from neighboring pixels
                float pixel = srcdata[xcoord + ycoord * x];
                if (pixel > high)
                    nearstrong = true;
            }
        }
        // WRITE to actual target pixel
        float current_pixel = srcdata[tidx + tidy * x];
        rdata[tidx + tidy * x] = current_pixel > high ? 1 : ((current_pixel > low && nearstrong) ? 1 : 0);

        
    }
}

std::shared_ptr<Image> CudaImage::edges(float low, float high, int mode)
{
    if (mode == 0) {
        validate();
        if (_type != floatgrayscale)
        {
            throw ImageException("Currently can only gradiant on floating point grayscale");
        }
        auto ret = std::make_shared<CudaImage>(_x, _y, _type);

        dim3 gridDim(_x/32, _y/32,1);
        dim3 blockDim(32, 32, 1);

        edges_kernel<<<gridDim, blockDim>>>((float*)_data, (float*)ret->_data, _x, _y, low, high);
       
        return ret;
    }
    return Image::edges(low, high);
}